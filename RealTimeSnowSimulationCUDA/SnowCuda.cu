#include "hip/hip_runtime.h"
﻿// Copyright SCIEMENT, Inc.
// by Hirofumi Seo, M.D., CEO & President

#include "Header/SnowCuda.h"
#include "header/Common.cuh"
#include "header/CountNeighbor.cuh"
#include "header/CohesionForce.cuh"
#include "header/InitParticle.cuh"
#include "header/Compression.cuh"
#include "header/MoveParticle.cuh"
#include "header/Collision.cuh"
#include "header/friction.cuh"
#include "header/FRNN.cuh"
#include "header/MoveInteractingCollider.cuh"

#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <hipcub/hipcub.hpp>




hipError_t SnowCUDA::initSnowCUDA(unsigned int _size, unsigned int _Max, float initPhaseSnow,float3 startFloor , float3 endTop ,std::string* error_message) {
    Max = _Max;
    size = _size;

    calcBin(startFloor, endTop);

    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        *error_message = "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_pos, Max * sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc pos failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_vel, Max * sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc vel failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_force, Max * sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc vel failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_radius, Max * sizeof(float));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc vel failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_pData, Max * sizeof(ParticleData));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc neighborCounts failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_debug, Max * sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc debug failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_colliderInteracting, MAX_OF_COLLIDER_INTERACTING * sizeof(Collider));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc Collider failed!";
        postErrorTask();
        return cuda_status;
    }
    cuda_status = hipMalloc((void**)&dev_colliderNotInteracting, MAX_OF_COLLIDER_NOT_INTERACTING * sizeof(Collider));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc Collider failed!";
        postErrorTask();
        return cuda_status;
    }
    cuda_status = hipMalloc((void**)&dev_colInterPos, MAX_OF_COLLIDER_INTERACTING * sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc Collider failed!";
        postErrorTask();
        return cuda_status;
    }
    cuda_status = hipMalloc((void**)&dev_colPos, MAX_OF_COLLIDER_NOT_INTERACTING * sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc Collider failed!";
        postErrorTask();
        return cuda_status;
    }
    cuda_status = hipMalloc((void**)&dev_preColPos, MAX_OF_COLLIDER_NOT_INTERACTING * sizeof(float3));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc Collider failed!";
        postErrorTask();
        return cuda_status;
    }
    cuda_status = hipMalloc((void**)&dev_PID, Max * sizeof(int));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc PID failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_BinID, Max * sizeof(int));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc BinID failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_Count, (Param.binLen + 1) * sizeof(int));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc binLen failed!";
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipMalloc((void**)&dev_PBM, (Param.binLen + 1) * sizeof(int));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc PBM failed!";
        postErrorTask();
        return cuda_status;
    }

    initParticles << <GRID_DIM, BLOCK_DIM >> > (dev_pData,dev_vel, dev_radius, initPhaseSnow, size, Param);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        *error_message = "Kernel initParticles launch failed: " + std::string(hipGetErrorString(cuda_status));
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "initParticles hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        postErrorTask();
        return cuda_status;
    }


}

hipError_t SnowCUDA::AddInteractingCollider(Collider* collider, float3* Colpos, std::string* error_message) {
    cuda_status = hipMemcpy(dev_colliderInteracting, collider, MAX_OF_COLLIDER_INTERACTING * sizeof(Collider), hipMemcpyHostToDevice);
    cuda_status = hipMemcpy(dev_colInterPos, Colpos, MAX_OF_COLLIDER_INTERACTING * sizeof(float3), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy density D2H failed!";
        postErrorTask();
        return cuda_status;
    }
    return cuda_status;
}

hipError_t SnowCUDA::AddCollider(Collider* collider, float3* Colpos, std::string* error_message) {
    cuda_status = hipMemcpy(dev_colliderNotInteracting, collider, MAX_OF_COLLIDER_NOT_INTERACTING * sizeof(Collider), hipMemcpyHostToDevice);
    cuda_status = hipMemcpy(dev_colPos, Colpos, MAX_OF_COLLIDER_NOT_INTERACTING * sizeof(float3), hipMemcpyHostToDevice);
    cuda_status = hipMemcpy(dev_preColPos, Colpos, MAX_OF_COLLIDER_NOT_INTERACTING * sizeof(float3), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy density D2H failed!";
        postErrorTask();
        return cuda_status;
    }
    return cuda_status;
}


hipError_t SnowCUDA::StartSimulation(float3* pos, std::string* error_message) {
    cuda_status = hipMemcpy(dev_pos, pos, Max * sizeof(float3), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy density D2H failed!";
        postErrorTask();
        return cuda_status;
    }
    initNeighbors << <GRID_DIM, BLOCK_DIM >> > (dev_pos, dev_pData, size);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        *error_message = "Kernel initNeighbors launch failed: " + std::string(hipGetErrorString(cuda_status));
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "initNeighbors hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        postErrorTask();
        return cuda_status;
    }

}


hipError_t SnowCUDA::UpdateCohesionForce(float3* debug, std::string* error_message) {

    if (isCrashed) return cuda_status;


    countNeighbors << <GRID_DIM, BLOCK_DIM >> > (dev_pos, dev_pData, dev_PID, dev_BinID, dev_PBM, size, Param);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        *error_message = "Kernel initParticles launch failed: " + std::string(hipGetErrorString(cuda_status));
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "initParticles hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        postErrorTask();
        return cuda_status;
    }


    getCohesionForce2 << <GRID_DIM, BLOCK_DIM >> > (dev_pos,dev_vel,dev_force, dev_pData, dev_PID,dev_BinID, dev_PBM ,dev_debug, size, Param);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        *error_message = "Kernel launch failed: " + std::string(hipGetErrorString(cuda_status));
        postErrorTask();
        return cuda_status;
    }

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        postErrorTask();
        return cuda_status;
    }



    cuda_status = hipMemcpy(debug, dev_debug, Max * sizeof(float3), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy debug D2H failed!";
        postErrorTask();
        return cuda_status;
    }

    return cuda_status;
}


hipError_t SnowCUDA::UpdateTransform(float3* pos, float* radius, float deltaTime,  std::string* error_message)
{
    if (isCrashed) return cuda_status;
    moveParticles << <GRID_DIM, BLOCK_DIM >> > (dev_pos,dev_vel, dev_force, dev_pData, deltaTime,dev_debug,size,Param);//커널 실행
    cuda_status = hipDeviceSynchronize();//디바이스 실행완료 기다리기

    compression << <GRID_DIM, BLOCK_DIM >> > (dev_pData, dev_radius, dev_debug, size, Param);//커널 실행(dev_pData는 디바이스에 있던 데이터 그대로 사용)
    cuda_status = hipDeviceSynchronize();//디바이스 실행완료 기다리기


    cuda_status = hipMemcpy(pos, dev_pos, Max * sizeof(float3), hipMemcpyDeviceToHost);//실행결과를 host로 출력
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy pos D2H failed!";
        postErrorTask();
        return cuda_status;
    }
    cuda_status = hipMemcpy(radius, dev_radius, Max * sizeof(float), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy radius D2H failed!";
        postErrorTask();
        return cuda_status;
    }

    return cuda_status;
}

hipError_t SnowCUDA::UpdateColliderPosition(float3* colInterPos, float deltaTime, std::string* error_message) {
    moveInteractingColliders << <1, MAX_OF_COLLIDER_INTERACTING >> > (dev_colliderInteracting, dev_colInterPos,deltaTime, dev_debug, Param);
    cuda_status = hipDeviceSynchronize();//디바이스 실행완료 기다리기
    cuda_status = hipMemcpy(colInterPos, dev_colInterPos, MAX_OF_COLLIDER_INTERACTING * sizeof(float3), hipMemcpyDeviceToHost);//collider들 위치 정보 cpu로 전송

    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy radius D2H failed!";
        postErrorTask();
        return cuda_status;
    }
    return cuda_status;
}

hipError_t SnowCUDA::UpdateCollision(float3* colPos, float3* debug,float dt, std::string* error_message) {
    if (isCrashed) return cuda_status;
    
    cuda_status = hipMemcpy(dev_colPos, colPos, MAX_OF_COLLIDER_NOT_INTERACTING * sizeof(float3), hipMemcpyHostToDevice);//device의 dev_collider업데이트
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy density D2H failed!";
        postErrorTask();
        return cuda_status;
    }


    calcColliderVelocity << <1, BLOCK_DIM >> > (dev_colPos, dev_preColPos, dev_colliderNotInteracting, Param, dt);
    cuda_status = hipDeviceSynchronize();



    //dev_colliderInteracting은 gpu메모리의 데이터 업데이트 없이 그대로 사용
    collide << <GRID_DIM, BLOCK_DIM >> > (
        dev_pos,
        dev_vel,
        dev_force,
        dev_colliderNotInteracting,
        dev_colliderInteracting,
        dev_colPos,
        dev_colInterPos,
        dev_pData,
        size, 
        Param
    );//충돌!
    cuda_status = hipDeviceSynchronize();//디바이스 실행완료 기다리기

    return cuda_status;
}


hipError_t SnowCUDA::UpdateFriction(float3* debug, std::string* error_message) {
    if (isCrashed) return cuda_status;
    friction << <GRID_DIM, BLOCK_DIM >> > (dev_pos, dev_vel, dev_force, dev_pData, dev_debug, size, Param);//커널 실행
    cuda_status = hipDeviceSynchronize();//디바이스 실행완료 기다리기
    cuda_status = hipMemcpy(debug, dev_debug, Max * sizeof(float3), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy density D2H failed!";
        postErrorTask();
        return cuda_status;
    }

    return cuda_status;
}


void SnowCUDA::postErrorTask() {
    hipFree(dev_pos);
    hipFree(dev_vel);
    hipFree(dev_pData);
    hipFree(dev_debug);
    hipFree(dev_colliderNotInteracting);
    hipFree(dev_colliderInteracting);
    hipFree(dev_PID);
    hipFree(dev_BinID);
    hipFree(dev_Count);
    hipFree(dev_PBM);

    isCrashed = true;
}

hipError_t SnowCUDA::Debug_GetParticleData(int index, ParticleData* particleData, float3* pos, float3* vel,  std::string* error_message) {
    if (isCrashed) return cuda_status;
    cuda_status = hipMemcpy(particleData, &dev_pData[index], sizeof(ParticleData), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy density D2H failed!";
        postErrorTask();
        return cuda_status;
    }
    cuda_status = hipMemcpy(pos, &dev_pos[index], sizeof(float3), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy density D2H failed!";
        postErrorTask();
        return cuda_status;
    }
    cuda_status = hipMemcpy(vel, &dev_vel[index], sizeof(float3), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy density D2H failed!";
        postErrorTask();
        return cuda_status;
    }
    return cuda_status;
}


hipError_t SnowCUDA::FRNN(std::string* error_message) {

    if (isCrashed) return cuda_status;

    resetPBM << <GRID_DIM, BLOCK_DIM >> > (dev_PBM, dev_Count, Param.binLen);

    getBinLoc << <GRID_DIM, BLOCK_DIM >> > (dev_pos, dev_BinID, dev_PID, size, Param);

    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        postErrorTask();
        return cuda_status;
    }

    thrust::device_ptr<int> dev_BinID_ptr(dev_BinID);
    thrust::device_ptr<int> dev_PID_ptr(dev_PID);

    thrust::sort_by_key(dev_BinID_ptr, dev_BinID_ptr + size, dev_PID_ptr);
    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        postErrorTask();
        return cuda_status;
    }

    countElem << <GRID_DIM, BLOCK_DIM >> > (dev_BinID, dev_Count, size);


    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        postErrorTask();
        return cuda_status;
    }


    void* d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, dev_Count, dev_PBM, Param.binLen + 1);
    // Scan 계산을 위한 메모리를 확인 

    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // 메모리 할당

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, dev_Count, dev_PBM, Param.binLen + 1);
    // Scan 계산

    hipFree(d_temp_storage);
    // Scan 계산용 임시 메모리 해제



    return cuda_status;

}

